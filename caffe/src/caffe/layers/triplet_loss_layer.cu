#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/triplet_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void TripletLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();

  caffe_gpu_sub(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
            diff_same_class_.mutable_gpu_data());
  caffe_gpu_sub(count, bottom[0]->gpu_data(), bottom[2]->gpu_data(),
            diff_diff_class_.mutable_gpu_data());

  Dtype loss = 0;
  Dtype pos_diff, neg_diff;
  for (int v = 0; v < batch_size_; ++v) {
    caffe_gpu_dot(vec_dimension_,
                  diff_same_class_.gpu_data() + v * vec_dimension_,
                  diff_same_class_.gpu_data() + v * vec_dimension_,
                  &pos_diff);
    caffe_gpu_dot(vec_dimension_,
                  diff_diff_class_.gpu_data() + v * vec_dimension_,
                  diff_diff_class_.gpu_data() + v * vec_dimension_,
		  &neg_diff);
    vec_loss_[v] = alpha_ + pos_diff - neg_diff;        
    vec_loss_[v] = std::max(Dtype(0), vec_loss_[v]);
    loss += vec_loss_[v];
  }

  loss /= batch_size_ * Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void TripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype scale = top[0]->cpu_diff()[0] / bottom[0]->num();
  const int n = bottom[0]->count();

  caffe_gpu_sub(n, diff_same_class_.gpu_data(), diff_diff_class_.gpu_data(),
            bottom[0]->mutable_gpu_diff());
  caffe_gpu_scal(n, scale, bottom[0]->mutable_gpu_diff());

  caffe_gpu_scale(n, -scale, diff_same_class_.gpu_data(),
                  bottom[1]->mutable_gpu_diff());

  caffe_gpu_scale(n, scale, diff_diff_class_.gpu_data(),
                  bottom[2]->mutable_gpu_diff());

  for (int v = 0; v < batch_size_; ++v) {
    if (vec_loss_[v] == 0) {
      caffe_gpu_set(vec_dimension_, Dtype(0),
                bottom[0]->mutable_gpu_diff() + v * vec_dimension_);
      caffe_gpu_set(vec_dimension_, Dtype(0),
                bottom[1]->mutable_gpu_diff() + v * vec_dimension_);
      caffe_gpu_set(vec_dimension_, Dtype(0),
                bottom[2]->mutable_gpu_diff() + v * vec_dimension_);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TripletLossLayer);

}  // namespace caffe
